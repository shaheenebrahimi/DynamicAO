#include "hip/hip_runtime.h"
#include "TanhActivation.cuh"
#include "NNException.h"
#include <iostream>

//__device__ float tanhActivation(float x) {
//	return 1.0f / (1 + exp(-x));
//}

__global__ void tanhActivationForward(float* Z, float* A,
										 int Z_x_dim, int Z_y_dim) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < Z_x_dim * Z_y_dim) {
		A[index] = tanh(Z[index]);
	}
}

//__global__ void tanhActivationBackprop(float* Z, float* dA, float* dZ,
//										  int Z_x_dim, int Z_y_dim) {
//
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (index < Z_x_dim * Z_y_dim) {
//		dZ[index] = dA[index] * tanh(Z[index]) * (1 - tanh(Z[index]));
//	}
//}

TanhActivation::TanhActivation(std::string name) {
	this->name = name;
}

TanhActivation::~TanhActivation()
{ }

Matrix& TanhActivation::forward(Matrix& Z) {
	this->Z = Z;
	A.allocateMemoryIfNotAllocated(Z.shape);

	dim3 block_size(256);
	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

	tanhActivationForward<<<num_of_blocks, block_size>>>(Z.data_device.get(), A.data_device.get(), Z.shape.x, Z.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform tanh forward propagation.");

	return A;
}

//Matrix& TanhActivation::backprop(Matrix& dA, float learning_rate) {
//	dZ.allocateMemoryIfNotAllocated(Z.shape);
//
//	dim3 block_size(256);
//	dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);
//	tanhActivationBackprop<<<num_of_blocks, block_size>>>(Z.data_device.get(), dA.data_device.get(),
//															 dZ.data_device.get(),
//															 Z.shape.x, Z.shape.y);
//	NNException::throwIfDeviceErrorsOccurred("Cannot perform tanh back propagation");
//
//	return dZ;
//}