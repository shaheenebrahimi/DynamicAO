#include "hip/hip_runtime.h"
#include "Evaluator.cuh"
#include "NNLayer.h"
#include "NNException.h"
#include "LinearLayer.cuh"
#include "TanhActivation.cuh"

#include <string>
#include <sstream>
#include <fstream>

using namespace std;

Evaluator::Evaluator(float learning_rate) :
	learning_rate(learning_rate)
{ }

Evaluator::Evaluator(const std::string &model) :
	learning_rate(0.01)
{
	loadEvaluator(model);
}

Evaluator::~Evaluator() {
	for (auto layer : layers) {
		delete layer;
	}
}

void Evaluator::addLayer(NNLayer* layer) {
	this->layers.push_back(layer);
}

Matrix Evaluator::forward(Matrix X) {
	Matrix Z = X;
	for (auto layer : layers) {
		Z = layer->forward(Z);
	}
	Y = Z;
	return Y;
}

Batch Evaluator::forwardBatch(Batch batchedX) {
	Batch batchedZ = batchedX;
	for (auto layer : layers) {
		batchedZ = layer->forwardBatch(batchedZ);
	}
	batchedY = batchedZ;
	return batchedY;
}

void Evaluator::loadEvaluator(const std::string& model) {
	ifstream in;
	in.open(model);
	if (!in.good()) {
	    cout << "Cannot read " << model << endl;
	    return;
	}
	//cout << "Loading " << model << endl;
	
	string line;
	stringstream ss;
	
	// Get meta data
	getline(in, line);
	ss = stringstream(line);
	int numLayers;
	ss >> numLayers;
	
	//cout << "layers: " << numLayers << endl;
	
	// Layer data
	for (int l = 0; l < numLayers; ++l) {
	    getline(in, line);
	    ss = stringstream(line);
	    // Get layer data
	    int inputs, outputs;
	    ss >> inputs; ss >> outputs;
		LinearLayer* layer = new LinearLayer("layer"+to_string(l), Shape(inputs, outputs));
	    //cout << "inputs: " << inputs << " outputs: " << outputs << endl;
	    // Get values
	    getline(in, line);
	    ss = stringstream(line);
	    layer->loadLayer(line);
	    layers.push_back(layer);
		if (l != numLayers - 1) layers.push_back(new TanhActivation("tanh" + to_string(l)));
	}
	in.close();
}

std::string Evaluator::getInfo() {
	std::string info = "";
	for (auto layer : layers) {
		info += layer->getName() + "\n";
	}
	return info;
}

float Evaluator::evaluate(const Matrix &input) {
	Matrix output = forward(input);
	output.copyDeviceToHost();
	return output[0];
}

std::vector<float> Evaluator::evaluateBatch(const Batch& input) {
	std::vector<float> res(input.batchSize);
	Batch output = forwardBatch(input);
	output.copyDeviceToHost();
	for (int b = 0; b < input.batchSize; ++b) {
		res[b] = output.data_host.get()[b];
	}
	return res;
}

void Evaluator::sharedBatchCompute(const Batch& input, struct hipGraphicsResource** outputResource)
{	
	float* output;
	hipGraphicsMapResources(1, outputResource, 0);

	size_t numBytes;
	hipGraphicsResourceGetMappedPointer((void**)&output, &numBytes, *outputResource);


	Batch res = forwardBatch(input);

	hipMemcpy(output, res.data_device.get(), numBytes, hipMemcpyDeviceToDevice);

	// unmap buffer object
	hipGraphicsUnmapResources(1, outputResource, 0);
}

//void Evaluator::backprop(Matrix predictions, Matrix target) {
//	dY.allocateMemoryIfNotAllocated(predictions.shape);
//	Matrix error = bce_cost.dCost(predictions, target, dY);
//
//	for (auto it = this->layers.rbegin(); it != this->layers.rend(); it++) {
//		error = (*it)->backprop(error, learning_rate);
//	}
//
//	hipDeviceSynchronize();
//}

std::vector<NNLayer*> Evaluator::getLayers() const {
	return layers;
}
